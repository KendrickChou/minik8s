#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define THREAD_NUM 256

#define MATRIX_SIZE 1000

const int blocks_num = (MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;

// CUDA 初始化
bool InitCUDA()
{
    int count;
    hipGetDeviceCount(&count);
    if (count == 0)
    {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for (i = 0; i < count; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if (hipGetDeviceProperties(&prop, i) == hipSuccess)
        {
            if (prop.major >= 1)
            {
                break;
            }
        }
    }

    if (i == count)
    {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);
    return true;
}

void generateMatrix(int *a, int size)
{
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            a[i * size + j] = rand();
        }
    }
}

__global__ static void addMatrixCUDA(const int *a, const int *b, int *c, int size)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int idx = bid * THREAD_NUM + tid;

    if (idx < size)
    {
        int max = size * size;
        for (int i = idx; i < max; i += size) {
            c[i] = a[i] + b[i];
        }
    }
}

int main()
{
    if (!InitCUDA())
        return 0;

    srand(0);

    int *a, *b, *c;
    a = (int *)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    b = (int *)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    c = (int *)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);

    generateMatrix(a, MATRIX_SIZE);
    generateMatrix(b, MATRIX_SIZE);

    int *cuda_a, *cuda_b, *cuda_c;

    hipMalloc((void **)&cuda_a, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    hipMalloc((void **)&cuda_b, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    hipMalloc((void **)&cuda_c, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);

    hipMemcpy(cuda_a, a, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice);

    addMatrixCUDA <<<blocks_num, THREAD_NUM, 0>>>(cuda_a, cuda_b, cuda_c, MATRIX_SIZE);

    hipMemcpy(c, cuda_c, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyDeviceToHost);

    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);

    return 0;
}
