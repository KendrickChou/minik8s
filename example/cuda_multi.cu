#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define THREAD_NUM 256

#define MATRIX_SIZE 1000

const int blocks_num = MATRIX_SIZE * (MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM + 1;

bool InitCUDA()
{
    int count;
    hipGetDeviceCount(&count);
    if (count == 0)
    {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for (i = 0; i < count; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if (hipGetDeviceProperties(&prop, i) == hipSuccess)
        {
            if (prop.major >= 1)
            {
                break;
            }
        }
    }

    if (i == count)
    {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);
    return true;
}

void generateMatrix(int *a, int size)
{
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            a[i * size + j] = rand();
        }
    }
}

__global__ static void multMatrixCUDA(const int *a, const int *b, int *c, int size)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int idx = bid * THREAD_NUM + tid;
    const int row = idx / size;
    const int column = idx % size;

    if (row < size && column < size)
    {
        int t = 0;

        for (int i = 0; i < size; i++)
        {
            t += a[row * size + i] * b[i * size + column];
        }
        c[row * size + column] = t;
    }
}

int main()
{
    if (!InitCUDA())
        return 0;

    srand(0);

    int *a, *b, *c;
    a = (int *)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    b = (int *)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    c = (int *)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);

    generateMatrix(a, MATRIX_SIZE);
    generateMatrix(b, MATRIX_SIZE);

    int *cuda_a, *cuda_b, *cuda_c;

    hipMalloc((void **)&cuda_a, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    hipMalloc((void **)&cuda_b, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
    hipMalloc((void **)&cuda_c, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);

    hipMemcpy(cuda_a, a, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice);

    multMatrixCUDA<<<blocks_num, THREAD_NUM, 0> > >(cuda_a, cuda_b, cuda_c, MATRIX_SIZE);

    hipMemcpy(c, cuda_c, sizeof(int) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyDeviceToHost);

    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);

    return 0;
}
